#include <iostream>
#include <stdio.h>
#include <stdlib.h>

#include <vector>
#include <string>
#include <hip/hip_runtime.h>

using namespace std;
__global__ void suma_vectores(float *c ,float *a , float *b,int N)
{
   int idx=blockIdx.x * blockDim.x+ threadIdx.x;
   if(idx<N)
   {
         c[idx]=a[idx] + b[idx];
   }
}
int main(void)
{
   float *a_h,*b_h,*c_h;
   float *a_d,*b_d,*c_d;
   int N=1000000;
   size_t size=N*sizeof(float);
   a_h = (float *) malloc (size);
   b_h = (float *) malloc (size);
   c_h = (float *) malloc (size);
   for (int i=0;i<N;i++)
   {
       a_h[i]=(float)i;
       b_h[i]=(float)(i+1);
   }
   hipMalloc((void**)& a_d,size);
   hipMalloc((void**)& b_d,size);
   hipMalloc((void**)& c_d,size);

   hipMemcpy(a_d,a_h,size,hipMemcpyHostToDevice);
   hipMemcpy(b_d,a_h,size,hipMemcpyHostToDevice);
   int block_size=8;
   int n_blocks=N/block_size + (N%block_size ==0 ? 0:1);
   suma_vectores <<< n_blocks,block_size >>> (c_d,a_d,b_d,N);
   hipMemcpy (c_h,c_d,size,hipMemcpyDeviceToHost);
   /*for (int i=0;i<N;i++)
   {
      cout<<c_h[i]<<" "<<endl;
   }*/

   free(a_h);
   free(b_h);
   free(c_h);
   return(0);


}