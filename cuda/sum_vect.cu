#include <iostream>
#include <stdio.h>
#include <stdlib.h>

#include <vector>
#include <string>
#include <hip/hip_runtime.h>

using namespace std;
__global__ void suma_vectores(float *c ,float *a , float *b,int N)
{
   int idx=blockIdx.x * blockDim.x+ threadIdx.x;
   if(idx<N)
   {
         c[idx]=a[idx] + b[idx];
   }
}

int main(void)
{
   float *a_h,*b_h,*c_h;
   float *a_d,*b_d,*c_d;
   int N=1000000;
   size_t size=N*sizeof(float);
   a_h = (float *) malloc (size);
   b_h = (float *) malloc (size);
   c_h = (float *) malloc (size);
   srand(1);

   for (int i = 0; i < N; ++i) {
    a_h[i] = rand();
    b_h[i] = rand();
   }


   hipMalloc((void**)& a_d,size);
   hipMalloc((void**)& b_d,size);
   hipMalloc((void**)& c_d,size);

   hipMemcpy(a_d,a_h,size,hipMemcpyHostToDevice);
   hipMemcpy(b_d,a_h,size,hipMemcpyHostToDevice);
   int block_size=256;
   int n_blocks=N/block_size + (N%block_size ==0 ? 0:1);
   suma_vectores <<< n_blocks,block_size >>> (c_d,a_d,b_d,N);
   hipMemcpy (c_h,c_d,size,hipMemcpyDeviceToHost);
   printf("Suma con  %d hebras con %d hebras por bloque!\n", N, block_size);
  
   /*for (int i=0;i<N;i++)
   {
      cout<<c_h[i]<<" "<<endl;
   }*/

   free(a_h);
   free(b_h);
   free(c_h);
   return(0);


}