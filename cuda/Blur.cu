
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <fstream>
#include <iostream>
#define BLUR_SIZE 3
using namespace std;

__global__
void blurKernel(float * in, float * out, int w, int h)
{
  int Col = blockIdx.x * blockDim.x + threadIdx.x;
  int Row = blockIdx.y * blockDim.y + threadIdx.y;
  if (Col < w && Row < h)
  {
    int pixVal = 0;
    int pixels = 0;
    // Get the average of the surrounding 2xBLUR_SIZE x 2xBLUR_SIZE box
    for(int blurRow = -BLUR_SIZE; blurRow < BLUR_SIZE+1; ++blurRow)
    {
      for(int blurCol = -BLUR_SIZE; blurCol < BLUR_SIZE+1; ++blurCol)
      {
        int curRow = Row + blurRow;
        int curCol = Col + blurCol;
        // Verify we have a valid image pixel
        if(curRow > -1 && curRow < h && curCol > -1 && curCol < w)
        {
          pixVal += in[curRow * w + curCol];
          pixels++; // Keep track of number of pixels in the accumulated total
        }
      }
    }
    // Write our new pixel value out
    out[Row * w + Col] = (float)(pixVal / pixels);
  }
}

void save_data(float r[225][225], float g[225][225], float b[225][225])
{
  ofstream archivo("bluur.dat");
  for (int i = 0; i < 225; ++i)
  {
    for (int j = 0; j < 225; ++j)
    {
          archivo<<r[i][j]<<" "<<g[i][j]<<" "<<b[i][j]<<" ";
    }
    archivo<<endl;
  }
}

void Blur(float r[225][225], float g[225][225], float b[225][225], int width, int height)
{
  float o_r[225][225];
  float o_g[225][225];
  float o_b[225][225];

  int size = width * height;
  int memSize = size * sizeof(float);

  float *d_A, *d_B;

  hipMalloc((void **) &d_A, memSize);
  hipMalloc((void **) &d_B, memSize);

  hipMemcpy(d_A, r, memSize, hipMemcpyHostToDevice);
  dim3 DimGrid(floor((width-1)/16 + 1), floor((height-1)/16+1), 1);
  dim3 DimBlock(16, 16, 1);
  blurKernel<<<DimGrid,DimBlock>>>(d_A, d_B, width, height);
    hipMemcpy(o_r, d_B, memSize, hipMemcpyDeviceToHost);

  hipMemcpy(d_A, g, memSize, hipMemcpyHostToDevice);

 
  blurKernel<<<DimGrid,DimBlock>>>(d_A, d_B, width, height);
  hipMemcpy(o_g, d_B, memSize, hipMemcpyDeviceToHost);

 
  hipMemcpy(d_A, b, memSize, hipMemcpyHostToDevice);

  blurKernel<<<DimGrid,DimBlock>>>(d_A, d_B, width, height);
  hipMemcpy(o_b, d_B, memSize, hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  save_data(o_r,o_g,o_b);
}

void leer_data(const char *file, float r[225][225], float g[225][225], float b[225][225])
{
  char buffer[100];
  ifstream archivo2("lena.dat");
  for (int ii = 0; ii < 225; ++ii)
  {
    for (int jj = 0; jj < 225; ++jj)
    {
          archivo2>>r[ii][jj]>>g[ii][jj]>>b[ii][jj];
    }
    archivo2.getline(buffer,100);
  }
}

int main()
{
  int width=225, height=225;
  float r[225][225];
  float g[225][225];
  float b[225][225];
  leer_data("lena.dat",r,g,b);
  Blur(r,g,b,width,height);
  return EXIT_SUCCESS;
}
