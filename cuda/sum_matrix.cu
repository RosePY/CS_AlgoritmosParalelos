#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stddef.h>



const int N = 16384;
const int THREADS_PER_BLOCK = 512;



__global__ void add_threads_blocks (int *a, int *b, int *c, int n) {

  int index = threadIdx.x * blockIdx.x * threadIdx.x;
  if (index < n) {
    c[index] = a[index] + b[index];
  }
}

int main(void) {
  int *a, *b, *c; 
  int *d_a, *d_b, *d_c; 
  size_t size = N * sizeof(int);

  srand(1);

  a = (int *) malloc(size);
  b = (int *) malloc(size);
  c = (int *) malloc(size);


  for (int i = 0; i < N; ++i) {
    a[i] = rand();
    b[i] = rand();
  }
  //uint kernelTime;
  //cutCreateTimer(&kernelTime);
  //cutResetTimer(kernelTime);
  hipMalloc((void **) &d_a, size);
  hipMalloc((void **) &d_b, size);
  hipMalloc((void **) &d_c, size);
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
  //cutStartTimer(kernelTime);
  add_threads_blocks<<<(N + (THREADS_PER_BLOCK - 1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_a, d_b, d_c, N);
  // cudaThreadSynchronize();
  //cutStopTimer(kernelTime);
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
  printf("Suma con  %d hebras con %d hebras por bloque!\n", N, THREADS_PER_BLOCK);
  //printf ("Time for the kernel: %f ms\n", cutGetTimerValue(kernelTime));
  free(a); free(b); free(c);
  hipFree(d_a); hipFree(d_b); hipFree(d_c);

  return 0;
}






