#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stddef.h>

/* Example from "Introduction to CUDA C" from NVIDIA website:
   https://developer.nvidia.com/cuda-education
   
   Compile with:
   $ nvcc example_intro.cu */

const int N = 16384;
const int THREADS_PER_BLOCK = 512;

/* Running one thread in each block (slides 24-32) */
__global__ void add_blocks (int *a, int *b, int *c) {
  /* blockIdx.x gives each block ID */
  c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

/* Running multiple threads in one block (slides 33-36) */
__global__ void add_threads (int *a, int *b, int *c) {
  /* threadIdx.x gives the thread ID in each block */
  c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

/* Running multiple threads in multiple blocks (slides 37-45).
   While doing this seems unecessary, in some cases we need threads
   since they have communication (__shared__ variables) and
   synchronization (__syncthreads()) mechanisms,
 */
__global__ void add_threads_blocks (int *a, int *b, int *c, int n) {
  /* blockDim.x gives the number of threads per block, combining it
     with threadIdx.x and blockIdx.x gives the index of each global
     thread in the device */
  int index = threadIdx.x * blockIdx.x * threadIdx.x;
  /* Typical problems are not friendly multiples of blockDim.x.
     Avoid accesing data beyond the end of the arrays */
  if (index < n) {
    c[index] = a[index] + b[index];
  }
}

int main(void) {
  int *a, *b, *c; /* Host (CPU) copies of a, b, c */
  int *d_a, *d_b, *d_c; /* Device (GPU) copies of a, b, c */
  size_t size = N * sizeof(int);

  srand(1);

  /* Allocate memory in device */
  hipMalloc((void **) &d_a, size);
  hipMalloc((void **) &d_b, size);
  hipMalloc((void **) &d_c, size);

  /* Allocate memory in host */
  a = (int *) malloc(size);
  b = (int *) malloc(size);
  c = (int *) malloc(size);

  /* Allocate random data in vectors a and b (inside host) */
  for (int i = 0; i < N; ++i) {
    a[i] = rand();
    b[i] = rand();
  }

  /* Copy data to device */
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);


  /* Launch add() kernel on device with N threads in N blocks */
  add_threads_blocks<<<(N + (THREADS_PER_BLOCK - 1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_a, d_b, d_c, N);
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

  /* Check if everything is alright */

  printf("Suma con  %d hebras!\n", N);

  /* Clean-up */
  free(a); free(b); free(c);
  hipFree(d_a); hipFree(d_b); hipFree(d_c);

  return 0;
}

/*
  /* Launch add() kernel on device with N blocks *
  add_blocks<<<N,1>>>(d_a, d_b, d_c);
  cudaMemcpy(c, d_c, size, cudaMemcpyDeviceToHost);

 /* /* Check if everything is alright *
  for (int i = 0; i < N; ++i) {
    assert(c[i] == a[i] + b[i]);
  }
  printf("Version with %d blocks executed succesfully!\n", N);

  /* Launch add() kernel on device with N threads *
  add_threads<<<1,N>>>(d_a, d_b, d_c);
  cudaMemcpy(c, d_c, size, cudaMemcpyDeviceToHost);

  /* Check if everything is alright *
  for (int i = 0; i < N; ++i) {
    assert(c[i] == a[i] + b[i]);
  }*
  printf("Suma con  %d hebras!\n", N);
  */