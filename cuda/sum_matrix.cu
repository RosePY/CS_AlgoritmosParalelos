#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stddef.h>



const int N = 16384;
const int THREADS_PER_BLOCK = 512;


__global__ void add_blocks (int *a, int *b, int *c) {

  c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}


__global__ void add_threads (int *a, int *b, int *c) {

  c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

__global__ void add_threads_blocks (int *a, int *b, int *c, int n) {

  int index = threadIdx.x * blockIdx.x * threadIdx.x;
  if (index < n) {
    c[index] = a[index] + b[index];
  }
}

int main(void) {
  int *a, *b, *c; 
  int *d_a, *d_b, *d_c; 
  size_t size = N * sizeof(int);

  srand(1);


  hipMalloc((void **) &d_a, size);
  hipMalloc((void **) &d_b, size);
  hipMalloc((void **) &d_c, size);


  a = (int *) malloc(size);
  b = (int *) malloc(size);
  c = (int *) malloc(size);


  for (int i = 0; i < N; ++i) {
    a[i] = rand();
    b[i] = rand();
  }


  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);


  add_threads_blocks<<<(N + (THREADS_PER_BLOCK - 1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_a, d_b, d_c, N);
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    printf("Suma con  %d hebras!\n", N);

  free(a); free(b); free(c);
  hipFree(d_a); hipFree(d_b); hipFree(d_c);

  return 0;
}



















/*
  /* Launch add() kernel on device with N blocks *
  add_blocks<<<N,1>>>(d_a, d_b, d_c);
  cudaMemcpy(c, d_c, size, cudaMemcpyDeviceToHost);

 /* /* Check if everything is alright *
  for (int i = 0; i < N; ++i) {
    assert(c[i] == a[i] + b[i]);
  }
  printf("Version with %d blocks executed succesfully!\n", N);

  /* Launch add() kernel on device with N threads *
  add_threads<<<1,N>>>(d_a, d_b, d_c);
  cudaMemcpy(c, d_c, size, cudaMemcpyDeviceToHost);

  /* Check if everything is alright *
  for (int i = 0; i < N; ++i) {
    assert(c[i] == a[i] + b[i]);
  }*
  printf("Suma con  %d hebras!\n", N);
  */