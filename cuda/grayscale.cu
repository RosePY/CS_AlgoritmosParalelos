
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <fstream>
#include <iostream>
#define CHANNELS 3 // we have 3 channels corresponding to RGB
using namespace std;

#define CHANNELS 3 // we have 3 channels corresponding to RGB
// The input image is encoded as unsigned characters [0, 255]
__global__ void colorConvert(float * Pout, float * Pin, int width, int height)
{
  int Col = threadIdx.x + blockIdx.x * blockDim.x;
  int Row = threadIdx.y + blockIdx.y * blockDim.y;
  if (Col < width && Row < height)
  {
    // get 1D coordinate for the grayscale image
    int greyOffset = Row*width + Col;
    // one can think of the RGB image having
    // CHANNEL times columns than the grayscale image
    int rgbOffset = greyOffset*CHANNELS;
    float r = Pin[rgbOffset]; // red value for pixel
    float g = Pin[rgbOffset + 1]; // green value for pixel
    float b = Pin[rgbOffset + 2]; // blue value for pixel
    // perform the rescaling and store it
    // We multiply by floating point constants
    Pout[greyOffset] = 0.21f*r + 0.71f*g + 0.07f*b;
  }
}


void save_data(float o[225][225])
{
  ofstream archivo("gray.dat");
  for (int i = 0; i < 225; ++i)
  {
    for (int j = 0; j < 225; ++j)
    {
          archivo<<o[i][j]<<" ";
    }
    archivo<<endl;
  }
}

void GrayScale(float m[225][225*3],int width, int height)
{
  float o[225][225];

  int size_in = width * (height*3);
  int size_out = width * height;
  int memSize_in = size_in * sizeof(float);
  int memSize_out = size_out * sizeof(float);

  float *d_A, *d_B;

  hipMalloc((void **) &d_A, memSize_in);
  hipMalloc((void **) &d_B, memSize_out);

  hipMemcpy(d_A, m, memSize_in, hipMemcpyHostToDevice);

  dim3 DimGrid(floor((width-1)/16 + 1), floor((height-1)/16+1), 1);
  dim3 DimBlock(16, 16, 1);
  colorConvert<<<DimGrid,DimBlock>>>(d_B, d_A, width, height);
 
  hipMemcpy(o, d_B, memSize_out, hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  save_data(o);
}

void leer_data(const char *file, float m[225][225*3])
{
  char buffer[100];
  ifstream archivo2("image.dat");
  for (int ii = 0; ii < 225; ++ii)
  {
    for (int jj = 0; jj < 225; ++jj)
    {
          archivo2>>m[ii][jj*3]>>m[ii][jj*3+1]>>m[ii][jj*3+2];
    }
    archivo2.getline(buffer,100);
  }
}

int main()
{
  int width=225, height=225;
  float m[225][225*3];
  leer_data("image.dat",m);
  GrayScale(m,width,height);

  return EXIT_SUCCESS;
}